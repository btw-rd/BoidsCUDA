#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <iostream>
#include <array>
#include <vector>
#include <chrono>
#include <random>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <glm.hpp>

#define f_separation 1.0f
#define f_alignment 1.0f
#define f_cohesion 1.0f
#define range 10.0f
#define velocity 2.0f
//#define N 1 << 12

struct Boid {
	glm::vec3 position{};
	glm::vec3 heading{};
};

__constant__ constexpr int N = 1 << 8;

///////////////////////////////
///		COPYING MEMORY IS TAKING FOREVER. DO MORE ON DEVICE EVEN IF IT'S SINGLE THREADED. BREAK OUT INTO MORE KERNELS, SORRY
////////////////////////////////


//__global__
//void update_distance(std::size_t i, const Boid* boids, float* distances)
//{
//	int index = threadIdx.x;
//	int stride = blockDim.x;
//
//	for (std::size_t j{ index }; j < N; j += stride)
//	{
//		//if (j == i) { continue; }
//
//		distances[j] = glm::distance(boids[i].position, boids[j].position);
//		//distances[j] = 68;
//	}
//}

//__global__
//void update_aggregates(std::size_t i, const Boid* boids, Boid* new_boids, float* distances, float* debug)
//{
//	//const float f_separation = 1;
//	//const float f_alignment = 1;
//	//const float f_cohesion = 1;
//	//const float range = 5;
//	//const float velocity = 2;
//
//	glm::vec3 new_heading(0);
//	glm::vec3 avg_heading(0);
//	glm::vec3 avg_position(0);
//	float neighbors = 0;
//
//	for (std::size_t j{ 0 }; j < N; ++j) {
//		//debug[j] = distances[j];
//		if (j == i) { continue; }
//
//		if (distances[j] < range) {
//			++neighbors;
//
//			//Influence from separation: normalized direction scaled by distance and force
//			if (distances[j] != 0) { new_heading += glm::normalize(boids[i].position - boids[j].position) * f_separation / distances[j]; }
//			//debug[j] = distances[j];
//
//			avg_heading += boids[j].heading - avg_heading / neighbors;
//
//			avg_position += boids[j].position - avg_position / neighbors;
//		}
//	}
//	glm::vec3 test_pos(glm::normalize(boids[i].position - boids[1].position) * f_separation / distances[1]);
//	//glm::vec3 test_pos = glm::vec3(0);
//
//	new_heading += avg_heading * f_alignment;
//	new_heading += (avg_position - boids[i].position) * f_cohesion;
//	new_heading = glm::normalize(new_heading);
//
//	new_boids[i] = Boid{ boids[i].position + new_heading * velocity, new_heading };
//	//new_boids[i] = Boid{ test_pos, boids[i].position + new_heading * velocity};
//}

__global__
void update(const Boid* boids, Boid* new_boids)
{
	int index = threadIdx.x;
	int stride = blockDim.x;

	for (std::size_t i{ index }; i < N; i += stride)
	{
		glm::vec3 new_heading(0);
		glm::vec3 avg_heading(0);
		glm::vec3 avg_position(0);
		float neighbors = 0;
		for (std::size_t j{ 0 }; j < N; ++j)
		{
			if (j == i) { continue; }

			float dis = glm::distance(boids[i].position, boids[j].position);
			
			if (dis < range && dis != 0)
			{
				++neighbors;

				new_heading += (boids[i].position - boids[j].position) * f_separation;

				avg_heading += boids[j].heading - avg_heading / neighbors;

				avg_position += boids[j].position - avg_position / neighbors;
			}
		}

		new_heading += avg_heading * f_alignment;
		new_heading += (avg_position - boids[i].position) * f_cohesion;
		new_heading = glm::normalize(new_heading);

		new_boids[i] = Boid{ boids[i].position + new_heading * velocity, new_heading };
	}
}

int main()
{
	//auto seed = std::random_device{}();
	//std::mt19937 mt{ seed };
	std::mt19937 mt{ 1 };
	std::uniform_int_distribution<> dist_pos{ 0, 200 };
	std::uniform_int_distribution<> dist_head{ -1, 1 };

	//std::vector<Boid> boids(N);
	//Boid* boids = new Boid[N];
	Boid* boids;
	Boid* new_boids;
	hipMallocManaged(&boids, N * sizeof(Boid));
	hipMallocManaged(&new_boids, N * sizeof(Boid));


	for (std::size_t i{ 0 }; i < N; ++i)
	{
		boids[i].position = glm::vec3({dist_pos(mt), dist_pos(mt), dist_pos(mt)});
		boids[i].heading = glm::normalize(glm::vec3(1));
	}

	auto start = std::chrono::high_resolution_clock::now();

	for (std::size_t i{ 0 }; i < 1; ++i)
	{
		update<<<1, 256>>>(boids, new_boids);
		hipDeviceSynchronize();

		std::swap(boids, new_boids);
		std::cout << "Position: " << boids[0].position.x << ", " << boids[0].position.y << ", " << boids[0].position.z << std::endl;
		std::cout << "Heading: " << boids[0].heading.x << ", " << boids[0].heading.y << ", " << boids[0].heading.z << std::endl;

	}

	auto stop = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
	std::cout << "Performance: " << duration.count() << "ms" << std::endl;

	//std::cout << "Heading: " << boids[0].heading.x << ", " << boids[0].heading.y << ", " << boids[0].heading.z << std::endl;
	//std::cout << "Position: " << boids[0].position.x << ", " << boids[0].position.y << ", " << boids[0].position.z << std::endl;

	hipFree(boids);
	hipFree(new_boids);

	return 0;
}
